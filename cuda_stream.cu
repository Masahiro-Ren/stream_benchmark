#include <hip/hip_runtime.h>
#include <iostream>

#define N (201326592)  // 768 MB per array (201,326,592 floats)
#define THREADS_PER_BLOCK 512
#define SCALAR 3.0f

__global__ void copyKernel(float *A, float *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        C[i] = A[i];
}

__global__ void scaleKernel(float *B, float *C, float scalar, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        B[i] = scalar * C[i];
}

__global__ void addKernel(float *A, float *B, float *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        C[i] = A[i] + B[i];
}

__global__ void triadKernel(float *A, float *B, float *C, float scalar, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        A[i] = B[i] + scalar * C[i];
}

void checkCuda(hipError_t result, const char *msg) {
    if (result != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " : " << hipGetErrorString(result) << std::endl;
        exit(EXIT_FAILURE);
    }
}

void reportTimeAndBandwidth(float ms, size_t bytes, const char* label) {
    double bandwidth = (bytes / (ms / 1000.0)) / (1 << 30);
    std::cout << label << ": " << ms << " ms, " << bandwidth << " GB/s" << std::endl;
}

int main() {

    float *d_A, *d_B, *d_C;
    float *h_A, *h_B, *h_C;
    size_t size = N * sizeof(float);
    size_t blocks = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    // Allocate host memory
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
        h_C[i] = 0.0f;
    }

    // Allocate device memory
    checkCuda(hipMalloc(&d_A, size), "hipMalloc A");
    checkCuda(hipMalloc(&d_B, size), "hipMalloc B");
    checkCuda(hipMalloc(&d_C, size), "hipMalloc C");

    // Copy initialized data to device
    checkCuda(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice), "Memcpy A");
    checkCuda(hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice), "Memcpy B");
    checkCuda(hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice), "Memcpy C");

    // Timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float msCopy, msScale, msAdd, msTriad;

    // Copy
    hipEventRecord(start);
    copyKernel<<<blocks, THREADS_PER_BLOCK>>>(d_A, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msCopy, start, stop);
    reportTimeAndBandwidth(msCopy, 2 * size, "Copy");

    // Scale
    hipEventRecord(start);
    scaleKernel<<<blocks, THREADS_PER_BLOCK>>>(d_B, d_C, SCALAR, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msScale, start, stop);
    reportTimeAndBandwidth(msScale, 2 * size, "Scale");

    // Add
    hipEventRecord(start);
    addKernel<<<blocks, THREADS_PER_BLOCK>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msAdd, start, stop);
    reportTimeAndBandwidth(msAdd, 3 * size, "Add");

    // Triad
    hipEventRecord(start);
    triadKernel<<<blocks, THREADS_PER_BLOCK>>>(d_A, d_B, d_C, SCALAR, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msTriad, start, stop);
    reportTimeAndBandwidth(msTriad, 3 * size, "Triad");

    // Clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
